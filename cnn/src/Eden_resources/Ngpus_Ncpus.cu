#include <Eden_resources/Ngpus_Ncpus.h>
#include <fmt/core.h>
#include <thread>

unsigned Eden_resources::get_gpus_count() {
    int count = 0;
    unsigned cuda_count = 0;
    
    hipGetDeviceCount(&count);
    if(count == 0) {
        fmt::print("There is no device.");
        return cuda_count;
    }
    for(int i = 0; i < count; ++i) {
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1) {
                ++cuda_count;
            }
            fmt::print("[{}] --{}", i, prop.name);
        }
    }
    if(cuda_count == 0) {
        fmt::print("There is no device supporting CUDA.");
    }
    return cuda_count;
}

unsigned Eden_resources::get_cpus_count() {
    const unsigned processor_count = std::thread::hardware_concurrency();
    if(processor_count == 0) {
        fmt::print("Found 0 cpus");
    }
    return processor_count;
}
