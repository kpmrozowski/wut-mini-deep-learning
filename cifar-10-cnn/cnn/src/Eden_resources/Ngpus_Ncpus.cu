#include <Eden_resources/Ngpus_Ncpus.h>
#include <fmt/core.h>
#include <thread>

unsigned Eden_resources::get_gpus_count() {
    int count = 0;
    unsigned cuda_count = 0;
    
    hipGetDeviceCount(&count);
    if(count == 0) {
        fmt::print("There is no device.\n");
        return cuda_count;
    }
    for(int i = 0; i < count; ++i) {
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1) {
                ++cuda_count;
            }
            fmt::print("[{}] --{}\n", i, prop.name);
        }
    }
    if(cuda_count == 0) {
        fmt::print("There is no device supporting CUDA.\n");
    }
    return cuda_count;
}

unsigned Eden_resources::get_cpus_count() {
    const unsigned processor_count = std::thread::hardware_concurrency();
    if(processor_count == 0) {
        fmt::print("Found 0 cpus\n");
    }
    return processor_count;
}
